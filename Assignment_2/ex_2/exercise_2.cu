
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <cmath>
#include <chrono>

#define ARRAY_SIZE 10000000
#define BLOCK_SIZE 256

__global__ void device_saxpy(float* x, float* y, const float a)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    y[i] = a * x[i] + y[i];
}

void host_saxpy(float x[], float y[], const float a)
{
    for(int i = 0; i < ARRAY_SIZE; i++) {
        y[i] = a * x[i] + y[i];
    }
}


int main()
{

    // ============= SET UP ARRAYS ============== //
    std::default_random_engine rdmGen;
    std::uniform_real_distribution<float> dist(0.0, 5.0);

    const float a = 1.0;

    float* x = (float*)malloc(ARRAY_SIZE * sizeof(float));
    float* y = (float*)malloc(ARRAY_SIZE * sizeof(float));

    for (int i = 0; i < ARRAY_SIZE; i++) {
        x[i] = dist(rdmGen);
        y[i] = dist(rdmGen);
    }



    // ============= START COMPUTING ON DEVICE ============== //
    printf("Computing SAXPY on the GPU... ");

    // Create, allocate and copy array to device
    auto start = std::chrono::system_clock::now();

    float* d_x = 0;
    float* d_y = 0;

    hipMalloc(&d_x, ARRAY_SIZE * sizeof(float));
    hipMalloc(&d_y, ARRAY_SIZE * sizeof(float));

    hipMemcpy(d_x, x, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);

    device_saxpy<<<(ARRAY_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE,
        BLOCK_SIZE>>>(d_x, d_y, a);
    hipDeviceSynchronize();

    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> host_time = (end-start) * 1000;

    printf("Done in %f ms!\n\n", host_time.count());




    // ============= START COMPUTING ON HOST ============== //
    printf("Computing SAXPY on the CPU... ");

    start = std::chrono::system_clock::now();
    host_saxpy(x, y, a);
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> device_time = (end-start) * 1000;
    
    printf("Done in %f ms!\n\n", device_time.count());




    // ============= COMPARE OUTPUTS ============== //

    // Get results from device and store in d_res
    hipMemcpy(x, d_y, ARRAY_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    printf("Comparing the output for each implementation... ");

    bool correct = true;
    for(int i = 0; i < ARRAY_SIZE; i++) {
        if(abs(x[i] - y[i]) > 0.0001) { // x is device result, y is host result
            correct = false;
            break;
        }
    }

    if(correct) printf("Correct!\n");
    else printf("Incorrect!\n");




    // ============= FREE RESOURCES ============== //
    free(y);
    free(x);

    hipFree(d_y);
    hipFree(d_x);

    return 0;
}