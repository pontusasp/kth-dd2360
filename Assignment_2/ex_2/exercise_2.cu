
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <cmath>
#include <chrono>

#define ARRAY_SIZE 1000000
#define BLOCK_SIZE 256

__global__ void device_saxpy(float* x, float* y, const float a)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    y[i] = a * x[i] + y[i];
}

void host_saxpy(float x[], float y[], const float a)
{
    for(int i = 0; i < ARRAY_SIZE; i++) {
        y[i] = a * x[i] + y[i];
    }
}


int main()
{
    std::default_random_engine rdmGen;
    std::uniform_real_distribution<float> dist(0.0, 5.0);

    const float a = 1.0;

    float* x = (float*)malloc(ARRAY_SIZE * sizeof(float));
    float* y = (float*)malloc(ARRAY_SIZE * sizeof(float));

    for (int i = 0; i < ARRAY_SIZE; i++) {
        x[i] = dist(rdmGen);
        y[i] = dist(rdmGen);
    }

    // Create, allocate and copy array to device
    float* d_x = 0;
    float* d_y = 0;

    hipMalloc(&d_x, ARRAY_SIZE * sizeof(float));
    hipMalloc(&d_y, ARRAY_SIZE * sizeof(float));

    hipMemcpy(d_x, x, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // call host func
    printf("Computing SAXPY on the CPU... ");

    auto start = std::chrono::system_clock::now();
    host_saxpy(x, y, a);
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> host_time = (end-start) * 1000;

    printf("Done in %f ms!\n\n", host_time.count());


    
    // call device func
    printf("Computing SAXPY on the GPU... ");

    start = std::chrono::system_clock::now();
    device_saxpy<<<(ARRAY_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE,
        BLOCK_SIZE>>>(d_x, d_y, a);
    hipDeviceSynchronize();
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> device_time = (end-start) * 1000;
    
    printf("Done in %f ms!\n\n", device_time.count());



    // Get results from device and store in d_res
    hipMemcpy(x, d_y, ARRAY_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // compare d_res to y
    printf("Comparing the output for each implementation... ");

    bool correct = true;
    for(int i = 0; i < ARRAY_SIZE; i++) {
        if(abs(x[i] - y[i]) > 0.0001) { // x is device result, y is host result
            correct = false;
            break;
        }
    }

    if(correct) printf("Correct!\n");
    else printf("Incorrect!\n");

    // Free up resources
    free(y);
    free(x);

    hipFree(d_y);
    hipFree(d_x);

    return 0;
}