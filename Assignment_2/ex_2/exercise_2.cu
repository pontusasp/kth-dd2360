
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <cmath>

#define ARRAY_SIZE 100000
#define BLOCK_SIZE 256

__global__ void device_saxpy(float* x, float* y, const float a)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    y[i] = a * x[i] + y[i];
}

void host_saxpy(float x[], float y[], const float a)
{
    for(int i = 0; i < ARRAY_SIZE; i++) {
        y[i] = a * x[i] + y[i];
    }
}


int main()
{
    std::default_random_engine rdmGen;
    std::uniform_real_distribution<float> dist(0.0, 5.0);

    const float a = 1.0;

    float x[ARRAY_SIZE];
    float y[ARRAY_SIZE];
    float d_res[ARRAY_SIZE];

    for (int i = 0; i < ARRAY_SIZE; i++) {
        x[i] = dist(rdmGen);
        y[i] = dist(rdmGen);
    }

    // Create, allocate and copy array to device
    float* d_x = 0;
    float* d_y = 0;

    hipMalloc(&d_x, ARRAY_SIZE * sizeof(float));
    hipMalloc(&d_y, ARRAY_SIZE * sizeof(float));

    hipMemcpy(d_x, &x, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, &y, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);

    printf("Computing SAXPY on the CPU... ");
    // call host func
    host_saxpy(x, y, a);
    printf("Done!\n\n");


    printf("Computing SAXPY on the GPU... ");
    // call device func
    device_saxpy<<<(ARRAY_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE,
        BLOCK_SIZE>>>(d_x, d_y, a);

    // Wait for device to finish
    hipDeviceSynchronize();
    printf("Done!\n\n");

    // Get results from device and store in d_res
    hipMemcpy(&d_res, d_y, ARRAY_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    printf("Comparing the output for each implementation... ");

    // compare d_res to y

    bool correct = true;

    for(int i = 0; i < ARRAY_SIZE; i++) {
        if(abs(d_res[i] - y[i]) > 0.0001) {
            correct = false;
            break;
        }
    }

    if(correct) printf("Correct!\n");
    else printf("Incorrect!\n");

    hipFree(d_y);
    hipFree(d_x);

    return 0;
}