
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <chrono>

#define BATCH_SIZE 20000

typedef struct
{
    float3 pos = {0.0};
    float3 vel = {0.0};
} Particle;

// Timestep for particles, f contains force to be applied to p.vel in x,y,z and w is time derivative
__global__ void device_timestep(Particle* p, float4 f)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float dt = f.w;
    
    // Update velocity
    p[i].vel.x = p[i].vel.x + f.x * dt;
    p[i].vel.y = p[i].vel.y + f.y * dt;
    p[i].vel.z = p[i].vel.z + f.z * dt;

    // Update position
    p[i].pos.x = p[i].pos.x + p[i].vel.x * dt;
    p[i].pos.y = p[i].pos.y + p[i].vel.y * dt;
    p[i].pos.z = p[i].pos.z + p[i].vel.z * dt;
}

void host_timestep(Particle* p, float4 f, const int num_particles)
{
    float dt = f.w;
    for(int i = 0; i < num_particles; i++) {
        // Update velocity
        p[i].vel.x = p[i].vel.x + f.x * dt;
        p[i].vel.y = p[i].vel.y + f.y * dt;
        p[i].vel.z = p[i].vel.z + f.z * dt;

        // Update position
        p[i].pos.x = p[i].pos.x + p[i].vel.x * dt;
        p[i].pos.y = p[i].pos.y + p[i].vel.y * dt;
        p[i].pos.z = p[i].pos.z + p[i].vel.z * dt;
    }
}

// returns if values successfully read or not.
bool setValuesFromArgs(int argc, char **argv, unsigned int *block_size, unsigned int *num_iterations, unsigned int *num_particles)
{
    if (argc < 4) {
        printf("Incorrect parameters!\nUsage: %s <block size> <num iterations>\
         <num particles> [1 extra arg for gpu benchmark output, 2 for cpu]\n", *argv);
        return false;
    }
    char *s;
    *block_size = strtoul(argv[1], &s, 10);
    *num_iterations = strtoul(argv[2], &s, 10);
    *num_particles = strtoul(argv[3], &s, 10);
    return true;
}

int main(int argc, char **argv)
{
    unsigned int block_size, num_iterations, num_particles;
    if(!setValuesFromArgs(argc, argv, &block_size, &num_iterations, &num_particles)) return 0;

    // Change num_threads to a multiple of block_size to prevent unexpected outcomes (memory size not matching up etc)
    num_particles = ((num_particles + block_size - 1) / block_size) * block_size; 

    bool gpuBench = argc == 5;
    bool cpuBench = argc == 6;

    if (!(gpuBench || cpuBench))
        printf("Starting simulation on %d particles with %d iterations, GPU set to use block size %d...\n\n", num_particles, num_iterations, block_size);
    
    Particle *particles = (Particle*)malloc(num_particles * sizeof(Particle));
    Particle *d_res;
    hipHostMalloc((void**)&d_res, num_particles * sizeof(Particle), hipHostMallocDefault, hipHostMallocDefault);

    std::default_random_engine rdmGen;
    std::uniform_real_distribution<float> posDist(-100.0, 100.0);
    std::uniform_real_distribution<float> velDist(-10.0, 10.0);

    for(int i = 0; i < num_particles; i++) {
        d_res[i].pos.x = particles[i].pos.x = posDist(rdmGen);
        d_res[i].pos.y = particles[i].pos.y = posDist(rdmGen);
        d_res[i].pos.z = particles[i].pos.z = posDist(rdmGen);

        d_res[i].vel.x = particles[i].vel.x = velDist(rdmGen);
        d_res[i].vel.y = particles[i].vel.y = velDist(rdmGen);
        d_res[i].vel.z = particles[i].vel.z = velDist(rdmGen);
    }

    float4 forces = {
         0.0,   // x
         0.0,   // y
        -9.82,  // z
         1.0    // dt
    };


    /*

    === Example ===
    ...
    int N = 3;
    int *arr, *d_arr;
    cudaMallocHost(&arr, N * sizeof(int));
    cudaMalloc(&d_arr, N * sizeof(int));

    cudaStream_t s_id;
    cudaStreamCreate(&s_id);

    cudaMemcpyAsync(d_arr, arr, N * sizeof(int), cudaMemcpyHostToDevice, s_id);

    // 3rd parameter is shared device memory
    fun<<<block_size, blocks, 0, s_id>>>;

    cudaStreamSynchronize(s_id);
    cudaMemcpyAsync(arr, d_arr, N * sizeof(int), cudaMemcpyDeviceToHost, s_id);

    cudaStreamDestroy(s_id);
    ...
    === = = = = ===


    === Lecture ===
    ...
    for (int i = 0; i < nStreams; i++) {

        int offset = i * streamSize;
        
        cudaMemcpyAsync(&d_a[offset], &a[offset], streamBytes, cudaMemcpyHostToDevice, stream[i]);

        kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);

        cudaMemcpyAsync(&a[offset], &d_a[offset], streamBytes, cudaMemcpyDeviceToHost, stream[i]);

    }
    ...
    === = = = = ===

    */


    // ============= START COMPUTING ON DEVICE ============== //
    if (!cpuBench) {
        if (!gpuBench)
            printf("Simulating on the GPU...\n");

        auto start1 = std::chrono::system_clock::now();
        
        // Create, allocate and copy array to device
        Particle* d_particles = 0;

        int num_streams = (num_particles + BATCH_SIZE - 1) / BATCH_SIZE;

        // Allocate memory for num_particles + potential padding
        hipMalloc(&d_particles, num_streams * BATCH_SIZE * sizeof(Particle));

        hipStream_t *s_id = (hipStream_t*)malloc(num_streams * sizeof(hipStream_t));

        for (int i = 0; i < num_streams; i++) {
            hipStreamCreate(&s_id[i]);
        }

        for(int i = 0; i < num_iterations; i++) {
            
            for(int j = 0; j < num_streams; j++) {
                int offset = j * BATCH_SIZE;

                hipMemcpyAsync(&d_particles[offset], &d_res[offset],
                    BATCH_SIZE * sizeof(Particle), hipMemcpyHostToDevice, s_id[j]);

                device_timestep<<<(BATCH_SIZE + block_size - 1) / block_size,
                    block_size, 0, s_id[j]>>>(d_particles, forces);
                    
                hipMemcpyAsync(&d_res[offset], &d_particles[offset],
                    num_particles * sizeof(Particle), hipMemcpyDeviceToHost, s_id[j]);
            }

            // Basically like cudaDeviceSynchronize, but using this to memorize stream synchronization
            for(int j = 0; j < num_streams; j++) {
                hipStreamSynchronize(s_id[j]);
            }

            // Ready to do stuff on host...
            
        }

        for (int i = 0; i < num_streams; i++) {
            hipStreamDestroy(s_id[i]);
        }

        hipFree(d_particles);

        auto end1 = std::chrono::system_clock::now();
        std::chrono::duration<double> device_time = end1-start1;
        
        if (!gpuBench)
            printf("\tDone in %f s!\n\n", device_time.count());
        else
            printf("%f\n", device_time.count());
    }


    if (!gpuBench) {
        // ============= START COMPUTING ON HOST ============== //
        if (!cpuBench)
            printf("Simulating on the CPU...\n");

        auto start2 = std::chrono::system_clock::now();

        for(int i = 0; i < num_iterations; i++) {
            host_timestep(particles, forces, num_particles);
        }

        auto end2 = std::chrono::system_clock::now();
        std::chrono::duration<double> host_time = end2-start2;

        if (!cpuBench) {
            printf("\tDone in %f s!\n\n", host_time.count());
            printf("All done!\n");
        }
        else printf("%f\n", host_time.count());
    }

    hipFree(d_res);
    free(particles);
    
    return 0;
}