
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <chrono>

typedef struct
{
    float3 pos = {0.0};
    float3 vel = {0.0};
} Particle;

// Timestep for particles, f contains force to be applied to p.vel in x,y,z and w is time derivative
__global__ void device_timestep(Particle* p, float4 f)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float dt = f.w;
    
    // Update velocity
    p[i].vel.x = p[i].vel.x + f.x * dt;
    p[i].vel.y = p[i].vel.y + f.y * dt;
    p[i].vel.z = p[i].vel.z + f.z * dt;

    // Update position
    p[i].pos.x = p[i].pos.x + p[i].vel.x * dt;
    p[i].pos.y = p[i].pos.y + p[i].vel.y * dt;
    p[i].pos.z = p[i].pos.z + p[i].vel.z * dt;
}

void host_timestep(Particle* p, float4 f, const int num_particles)
{
    float dt = f.w;
    for(int i = 0; i < num_particles; i++) {
        // Update velocity
        p[i].vel.x = p[i].vel.x + f.x * dt;
        p[i].vel.y = p[i].vel.y + f.y * dt;
        p[i].vel.z = p[i].vel.z + f.z * dt;

        // Update position
        p[i].pos.x = p[i].pos.x + p[i].vel.x * dt;
        p[i].pos.y = p[i].pos.y + p[i].vel.y * dt;
        p[i].pos.z = p[i].pos.z + p[i].vel.z * dt;
    }
}

// returns if values successfully read or not.
bool setValuesFromArgs(int argc, char **argv, unsigned int *block_size, unsigned int *num_iterations, unsigned int *num_particles)
{
    if (argc < 4) {
        printf("Incorrect parameters!\nUsage: %s <block size> <num iterations>\
         <num particles> [1 extra arg for gpu benchmark output, 2 for cpu]\n", *argv);
        return false;
    }
    char *s;
    *block_size = strtoul(argv[1], &s, 10);
    *num_iterations = strtoul(argv[2], &s, 10);
    *num_particles = strtoul(argv[3], &s, 10);
    return true;
}

int main(int argc, char **argv)
{
    unsigned int block_size, num_iterations, num_particles;
    if(!setValuesFromArgs(argc, argv, &block_size, &num_iterations, &num_particles)) return 0;

    // Change num_threads to a multiple of block_size to prevent unexpected outcomes (memory size not matching up etc)
    num_particles = ((num_particles + block_size - 1) / block_size) * block_size; 

    bool gpuBench = argc == 5;
    bool cpuBench = argc == 6;

    if (!(gpuBench || cpuBench))
        printf("Starting simulation on %d particles with %d iterations, GPU set to use block size %d...\n\n", num_particles, num_iterations, block_size);
    
    Particle *particles = (Particle*)malloc(num_particles * sizeof(Particle));
    Particle *d_res;
    hipHostMalloc((void**)&d_res, num_particles * sizeof(Particle), hipHostMallocDefault, hipHostMallocDefault);

    std::default_random_engine rdmGen;
    std::uniform_real_distribution<float> posDist(-100.0, 100.0);
    std::uniform_real_distribution<float> velDist(-10.0, 10.0);

    for(int i = 0; i < num_particles; i++) {
        particles[i].pos.x = posDist(rdmGen);
        particles[i].pos.y = posDist(rdmGen);
        particles[i].pos.z = posDist(rdmGen);

        particles[i].vel.x = velDist(rdmGen);
        particles[i].vel.y = velDist(rdmGen);
        particles[i].vel.z = velDist(rdmGen);
    }

    float4 forces = {
         0.0,   // x
         0.0,   // y
        -9.82,  // z
         1.0    // dt
    };



    // ============= START COMPUTING ON DEVICE ============== //
    if (!cpuBench) {
        if (!gpuBench)
            printf("Simulating on the GPU...\n");

        auto start1 = std::chrono::system_clock::now();
        
        // Create, allocate and copy array to device
        Particle* d_particles = 0;
        hipMalloc(&d_particles, num_particles * sizeof(Particle));

        for(int i = 0; i < num_iterations; i++) {
            hipMemcpy(d_particles, particles, num_particles * sizeof(Particle), hipMemcpyHostToDevice);

            device_timestep<<<(num_particles + block_size - 1) / block_size,
                block_size>>>(d_particles, forces);
                
            hipDeviceSynchronize();
            hipMemcpy(d_res, d_particles, num_particles * sizeof(Particle), hipMemcpyDeviceToHost);
        }

        hipFree(d_particles);

        auto end1 = std::chrono::system_clock::now();
        std::chrono::duration<double> device_time = end1-start1;
        
        if (!gpuBench)
            printf("\tDone in %f s!\n\n", device_time.count());
        else
            printf("%f\n", device_time.count());
    }


    if (!gpuBench) {
        // ============= START COMPUTING ON HOST ============== //
        if (!cpuBench)
            printf("Simulating on the CPU...\n");

        auto start2 = std::chrono::system_clock::now();

        for(int i = 0; i < num_iterations; i++) {
            host_timestep(particles, forces, num_particles);
        }

        auto end2 = std::chrono::system_clock::now();
        std::chrono::duration<double> host_time = end2-start2;

        if (!cpuBench) {
            printf("\tDone in %f s!\n\n", host_time.count());
            printf("All done!\n");
        }
        else printf("%f\n", host_time.count());
    }

    hipFree(d_res);
    free(particles);
    
    return 0;
}